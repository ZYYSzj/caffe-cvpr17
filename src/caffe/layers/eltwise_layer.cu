#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <stdio.h>
#include "caffe/layers/eltwise_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxForward(const int nthreads, const Dtype* bottom_data_a,
    const Dtype* bottom_data_b, const int blob_idx, Dtype* top_data,
    int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    if (bottom_data_a[index] > bottom_data_b[index]) {
      // only update for very first bottom_data blob (blob_idx == 0)
      if (blob_idx == 0) {
        maxval = bottom_data_a[index];
        top_data[index] = maxval;
        maxidx = blob_idx;
        mask[index] = maxidx;
      }
    } else {
      maxval = bottom_data_b[index];
      top_data[index] = maxval;
      maxidx = blob_idx + 1;
      mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void ProdForward(const int nthreads, const Dtype* bottom_data_a, const Dtype* bottom_data_b, const int dim, const int channels, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
   int num = index / (dim * channels); 
   int c = (index / dim) % channels;
   int id = index % dim;
   top_data[(num * channels + c) * dim + id] = bottom_data_a[(num * channels + c) * dim + id] * bottom_data_b[num * dim + id];
  }
}

template <typename Dtype>
void EltwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int* mask = NULL;
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    if (cross_channel_) {
      const int dim = top[0]->count(2);
      const int channels = top[0]->channels();
      ProdForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), dim, channels, top_data);
      for (int i = 2; i < bottom.size(); ++i) {
	ProdForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data, bottom[i]->gpu_data(), dim, channels, top_data);
      }
    } else {
      caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
	  top_data);
      for (int i = 2; i < bottom.size(); ++i) {
	caffe_gpu_mul(count, top_data, bottom[i]->gpu_data(), top_data);
      }
    }
    //for (int n = 0; n < bottom[0]->num(); n++) {
    //  for (int c = 0; c < bottom[0]->channels(); c++) {
    //    const Dtype* cur_b0 = bottom[0]->cpu_data() + bottom[0]->offset(n, c);
    //    const Dtype* cur_b1 = bottom[1]->cpu_data() + bottom[1]->offset(n);
    //    const Dtype* cur_b2 = bottom[2]->cpu_data() + bottom[2]->offset(n);
    //    const Dtype* cur_top = top[0]->cpu_data() + top[0]->offset(n, c);
    //    printf("n = %d, c = %d\n", n, c);
    //    for (int i = 0; i < bottom[0]->count(2); i++) {
    //      printf("bottom0: %f, bottom1: %f, bottom2: %f\n top: %f \n", float(cur_b0[i]), float(cur_b1[i]), float(cur_b2[i]), float(cur_top[i]));
    //    }
    //  }
    //}

    break;
  case EltwiseParameter_EltwiseOp_SUM:
    caffe_gpu_set(count, Dtype(0.), top_data);
    // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
    for (int i = 0; i < bottom.size(); ++i) {
      caffe_gpu_axpy(count, coeffs_[i], bottom[i]->gpu_data(), top_data);
    }
    break;
  case EltwiseParameter_EltwiseOp_MAX:
    mask = max_idx_.mutable_gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), 0, top_data, mask);
    for (int i = 2; i < bottom.size(); ++i) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      MaxForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_data, bottom[i]->gpu_data(), i-1, top_data, mask);
    }
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}

template <typename Dtype>
__global__ void MaxBackward(const int nthreads, const Dtype* top_diff,
    const int blob_idx, const int* mask, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype gradient = 0;
    if (mask[index] == blob_idx) {
      gradient += top_diff[index];
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void ProdBackwardStable(const int nthreads, const Dtype* top_diff, const Dtype* temp_diff, const Dtype* bottom_a_data, const int channels, const int dim, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
   int num = index / dim;
   int id = index % dim;
   const Dtype* cur_top_diff = top_diff + num * channels * dim +id;
   const Dtype* cur_temp_diff = temp_diff + num * dim + id;
   const Dtype* cur_bottom_a_data = bottom_a_data + num * channels * dim + id;
   Dtype* cur_bottom_diff = bottom_diff + num * dim + id;
   Dtype diff = 0;
   for (int i = 0; i < channels; i++) {
     diff += cur_top_diff[i * dim] * cur_bottom_a_data[i * dim];
   }
   cur_bottom_diff[0] = diff * cur_temp_diff[0];
  }
}

template <typename Dtype>
__global__ void ProdBackward(const int nthreads, const Dtype* top_data, const Dtype* top_diff, const Dtype* bottom_data, const int channels, const int dim, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
   int num = index / dim;
   int id = index % dim;
   const Dtype* cur_top_diff = top_diff + num * channels * dim +id;
   const Dtype* cur_top_data = top_data + num * channels * dim +id;
   const Dtype* cur_bottom_data = bottom_data + num * dim + id;
   Dtype* cur_bottom_diff = bottom_diff + num * dim + id;
   Dtype diff = 0;
   for (int i = 0; i < channels; i++) {
     diff += cur_top_diff[i * dim] * cur_top_data[i * dim] / cur_bottom_data[0]; 
   }
   cur_bottom_diff[0] = diff;
  }
}

template <typename Dtype>
void EltwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int* mask = NULL;
  const int count = top[0]->count();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  for (int i = 0; i < bottom.size(); ++i) {
    if (propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        if (cross_channel_) {
	  const int dim = top[0]->count(2);
	  const int channels = top[0]->channels();
	  const int reference_count = bottom[1]->count();
	  Blob<Dtype> temp_blob(top[0]->num(), 1, top[0]->height(), top[0]->width());
	  Dtype* temp_blob_diff = temp_blob.mutable_gpu_diff();
	  caffe_gpu_set(reference_count, Dtype(1), temp_blob_diff);
	  if (stable_prod_grad_) {
	    bool initialized = false;
	    for (int j = 1; j < bottom.size(); ++j) {
	      if (i == j) { continue; }
	      if (!initialized) {
		caffe_copy(reference_count, bottom[j]->gpu_data(), temp_blob_diff);
		initialized = true;
	      } else {
		caffe_gpu_mul(reference_count, temp_blob_diff, bottom[j]->gpu_data(), temp_blob_diff);
	      }
	    }
	    if (i ==0) {
	      ProdForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, temp_blob_diff, dim, channels, bottom_diff);
	    } else {
	      const Dtype* bottom_a_data = bottom[0]->gpu_data();
	      ProdBackwardStable<Dtype><<<CAFFE_GET_BLOCKS(reference_count), CAFFE_CUDA_NUM_THREADS>>>(reference_count, top_diff, temp_blob_diff, bottom_a_data, channels, dim, bottom_diff);
	    }
	  } else {
	    if (i == 0) {
	      caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
	      caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
	    } else {
	      const Dtype* bottom_a_data = bottom[0]->gpu_data();
	      ProdBackward<Dtype><<<CAFFE_GET_BLOCKS(reference_count), CAFFE_CUDA_NUM_THREADS>>>(reference_count, top_data, top_diff, bottom_data, channels, dim, bottom_diff);
	    }
	   
	  }
	} else {
	  if (stable_prod_grad_) {
	    bool initialized = false;
	    for (int j = 0; j < bottom.size(); ++j) {
	      if (i == j) { continue; }
	      if (!initialized) {
		caffe_copy(count, bottom[j]->gpu_data(), bottom_diff);
		initialized = true;
	      } else {
		caffe_gpu_mul(count, bottom[j]->gpu_data(), bottom_diff,
		    bottom_diff);
	      }
	    }
	  } else {
	    caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
	  }
	  caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
	}
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        if (coeffs_[i] == Dtype(1.)) {
          caffe_copy(count, top_diff, bottom_diff);
        } else {
          caffe_gpu_scale(count, coeffs_[i], top_diff, bottom_diff);
        }
        break;
      case EltwiseParameter_EltwiseOp_MAX:
        mask = max_idx_.gpu_data();
        MaxBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, i, mask, bottom_diff);
        break;
      default:
        LOG(FATAL) << "Unknown elementwise operation.";
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EltwiseLayer);

}  // namespace caffe
