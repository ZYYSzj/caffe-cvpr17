#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/overlap_accuracy_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void OverlapAccuracyForwardGPU(const int nthreads, const int dim, const Dtype* prediction, const Dtype* label, Dtype* pre, Dtype* recall) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const Dtype* cur_prediction = prediction + index * dim;
    const Dtype* cur_label = label + index * dim;
    Dtype count_pre, count_gt, count_it;
    count_pre = 0; count_gt = 0; count_it = 0;
    for (int i = 0; i < dim; i++) {
      if(cur_label[i] > 0.5 && cur_prediction[i] >= 0.5) {
	count_pre++;
	count_gt++;
	count_it++;
      } else if (label[i] >0.5) {
	count_gt++;
      } else if (cur_prediction[i] >= 0.5) {
	count_pre++;
      }
    }
    pre[index] = count_it / (count_pre + Dtype(FLT_MIN));
    recall[index] = count_it / (count_gt + Dtype(FLT_MIN));
  }
}

template <typename Dtype>
void OverlapAccuracyLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  int num = bottom[0]->num();
  int dim = bottom[0]->count(1);
  Dtype* pre_data = pre_.mutable_gpu_data();
  Dtype* recall_data = recall_.mutable_gpu_data();
  const Dtype* prediction = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data(); 
  OverlapAccuracyForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(num),
      CAFFE_CUDA_NUM_THREADS>>>(num, dim, prediction, label, pre_data, recall_data);
  Dtype P, R;
  caffe_gpu_asum(num, pre_data, &P);
  caffe_gpu_asum(num, recall_data, &R);
  top[0]->mutable_cpu_data()[0] = P / num;
  if (top.size() >= 2) {
    top[1]->mutable_cpu_data()[0] = R / num;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(OverlapAccuracyLayer);

}  // namespace caffe
