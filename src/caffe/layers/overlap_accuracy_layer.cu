#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/overlap_accuracy_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void OverlapAccuracyForwardGPU(const int nthreads, const Dtype* prediction, const Dtype* label, Dtype* intersection, Dtype* union_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    if(label[index] > 0.5 && prediction[index] >= 0.5) {
      intersection[index] = 1;
      union_data[index] = 1;
    } else if (label[index] >0.5 || prediction[index] >= 0.5) {
      intersection[index] = 0;
      union_data[index] = 1;
    } else {
      intersection[index] = 0;
      union_data[index] = 0;
    }
  }
}

template <typename Dtype>
void OverlapAccuracyLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  int num = bottom[0]->num();
  Dtype* intersection_data = intersection_.mutable_gpu_data();
  Dtype* union_data = union_.mutable_gpu_data();
  const Dtype* prediction = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data(); 
  OverlapAccuracyForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, prediction, label, intersection_data, union_data);
  Dtype I, U, accuracy;
  caffe_gpu_asum(count, intersection_data, &I);
  caffe_gpu_asum(count, union_data, &U);
  accuracy = I / U ;
  top[0]->mutable_cpu_data()[0] = accuracy;
}

INSTANTIATE_LAYER_GPU_FUNCS(OverlapAccuracyLayer);

}  // namespace caffe
