#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/smooth_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "iostream"
#include "stdio.h"

namespace caffe {


  template <typename Dtype>
    __global__ void SmoothPoolForward(const int nthreads, const Dtype* bottom_data, const int num, const int channels, const int dim, int* index_data, Dtype* value_data, const bool unique_smooth, const bool has_smooth_blobs, const Dtype z, const Dtype* smooth_data, Dtype* weight, Dtype* w_norm_data, Dtype* top_data) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const int n = index / channels;
	const int c = index % channels;
	int *U, *G, *L, *ind_tmp;
	U = index_data + 3 * dim * index;
	G = U + dim;
	L = G + dim;
	const Dtype* cur_bottom = bottom_data + dim * index;
	Dtype* v_tmp = value_data + dim * index;
	Dtype* w = weight + dim * index;
	Dtype* w_norm = w_norm_data + index;
	Dtype* o = top_data + index;
	double theta,  w_tmp;
	Dtype mu;
	if (has_smooth_blobs) {
	  if (unique_smooth) {
	    mu = smooth_data[0];
	  } else {
	    mu = smooth_data[c];
	  }
	} else {
	  if (unique_smooth) {
	    mu = smooth_data[n];
	  } else {
	    mu = smooth_data[index];
	  }
	}

	for (int i = 0; i < dim; i++) {
	  v_tmp[i] = Dtype(1) / (mu + Dtype(FLT_MIN)) * cur_bottom[i];
	  U[i] = i;
	}
	double s = 0, ds = 0, ro = 0, dro = 0;
	int n_U, n_G, n_L; 
	n_U = dim;
	while (n_U > Dtype(0)) {
	  int k = n_U-1;
	  n_G = 0; n_L =0;
	  ds = 0;
	  for(int i = 0; i < n_U; i++) {
	    if (v_tmp[U[i]] >= v_tmp[U[k]]) {
	      G[n_G++] = U[i];
	      ds += double(v_tmp[U[i]]);
	    } else {
	      L[n_L++] = U[i];
	    }
	  }
	  dro = double(n_G);

	  if ((s+ds) -(ro + dro) * double(v_tmp[U[k]]) < z) {
	    s += ds; ro += dro;
	    ind_tmp = U;
	    U = L;
	    n_U = n_L;
	    L = ind_tmp;
	  } else {
	    ind_tmp = U;
	    U = G;
	    n_U = n_G -1;
	    G = ind_tmp;
	  }
	}
	theta = (s-double(z)) / (ro + DBL_MIN);

	o[0] = Dtype(0);
	w_norm[0] = 0;
	for (int i = 0; i < dim; i++) {
	  w_tmp = double(v_tmp[i]) - theta;
	  w_tmp = w_tmp > 0 ? w_tmp : double(0);
	  w[i] = Dtype(w_tmp);
	  w_norm[0] += w[i] * w[i];
	  o[0] += cur_bottom[i] * Dtype(w[i]);
	}
	o[0] -= Dtype(0.5) * mu * w_norm[0];
      }
    }



  template <typename Dtype>
    void SmoothPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      int count = top[0]->count();
      const Dtype* smooth_data = smooth_->gpu_data();
      Dtype* weight_data =  weight_.mutable_gpu_data();
      Dtype* w_norm_data = w_norm_.mutable_gpu_data();
      Blob<int> index_set(3*num_, channels_, height_, width_);
      int* index_data = index_set.mutable_gpu_data();
      Blob<Dtype> value(num_, channels_, height_, width_);
      Dtype* value_data = value.mutable_gpu_data();
      SmoothPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, num_, channels_, dim_, 
	  index_data, value_data, unique_smooth_, has_smooth_blobs_, z_, smooth_data, weight_data, w_norm_data, top_data);
      CUDA_POST_KERNEL_CHECK;
    }


  //
  //
  //template <typename Dtype>
  //__global__ void AvePoolBackward(const int nthreads, const Dtype* const top_diff, const int num, const int channels, const int height,
  //                                const int width, const int pooled_height, const int pooled_width,
  //                                const int kernel_h, const int kernel_w, const int stride_h,
  //                                const int stride_w, const int pad_h, const int pad_w,
  //                                Dtype* const bottom_diff) {
  //                                  CUDA_KERNEL_LOOP(index, nthreads) {
  //                                    // find out the local index
  //                                    // find out the local offset
  //                                    const int w = index % width + pad_w;
  //                                    const int h = (index / width) % height + pad_h;
  //                                    const int c = (index / width / height) % channels;
  //                                    const int n = index / width / height / channels;
  //                                    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
  //                                    const int phend = min(h / stride_h + 1, pooled_height);
  //                                    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
  //                                    const int pwend = min(w / stride_w + 1, pooled_width);
  //                                    Dtype gradient = 0;
  //                                    const Dtype* const top_diff_slice =
  //                                    top_diff + (n * channels + c) * pooled_height * pooled_width;
  //                                    for (int ph = phstart; ph < phend; ++ph) {
  //                                      for (int pw = pwstart; pw < pwend; ++pw) {
  //                                        // figure out the pooling size
  //                                        int hstart = ph * stride_h - pad_h;
  //                                        int wstart = pw * stride_w - pad_w;
  //                                        int hend = min(hstart + kernel_h, height + pad_h);
  //                                        int wend = min(wstart + kernel_w, width + pad_w);
  //                                        int pool_size = (hend - hstart) * (wend - wstart);
  //                                        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
  //                                      }
  //                                    }
  //                                    bottom_diff[index] = gradient;
  //                                  }
  //                                }

  template <typename Dtype>
    __global__ void SmoothPoolBackward(const int nthreads, const Dtype* top_diff, const int num, const int channels, const Dtype* w_norm_data, Dtype* smooth_diff) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const Dtype* cur_top_diff = top_diff + index;
	const Dtype* cur_w_norm = w_norm_data + index;

	for (int i = 0; i < num; i++) {
	  smooth_diff[index] += cur_top_diff[i*channels] * cur_w_norm[i*channels];
	}
	smooth_diff[index] *= -Dtype(0.5); 
      }
    }

  template <typename Dtype>
    __global__ void caffe_gpu_hadamard_product(const int nthreads, const Dtype alpha, const Dtype* a, const Dtype* b, Dtype* c) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	c[index] = alpha * a[index] * b[index];
      }
    }



  template <typename Dtype>
    void SmoothPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      LOG(INFO) << "start backward_gpu";
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* weight_data = weight_.gpu_data();
      const Dtype* w_norm_data = w_norm_.gpu_data();
      if (propagate_down[0]) {
	//Gradient with respect to bottom [0]
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	const Dtype* top_cpu_diff = top[0]->cpu_diff();
	caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
	for (int n = 0; n < num_; n++) {
	  for (int c = 0; c < channels_; c++) {
	    const Dtype* cur_weight_data = weight_data + weight_.offset(n, c);
	    Dtype* cur_bottom_diff = bottom_diff + bottom[0]->offset(n, c);
	    const Dtype* cur_top_diff = top_cpu_diff + top[0]->offset(n, c); 
	    caffe_gpu_axpy(dim_, cur_top_diff[0], cur_weight_data,  cur_bottom_diff);
	  }
	}
      }
      if (!has_smooth_blobs_ && propagate_down[1]) {
	Dtype* smooth_diff = smooth_->mutable_gpu_diff();
	Dtype* smooth_cpu_diff = smooth_->mutable_cpu_diff();
	caffe_gpu_set(smooth_->count(), Dtype(0), smooth_diff);
	if (unique_smooth_) {
	  for (int n = 0; n < num_; n++) {
	    Dtype* cur_smooth_diff = smooth_cpu_diff + n;
	    const Dtype* cur_w_norm = w_norm_data + w_norm_.offset(n);
	    const Dtype* cur_top_diff = top_diff + top[0]->offset(n);
	    caffe_gpu_dot(channels_, cur_w_norm, cur_top_diff, cur_smooth_diff);
	    cur_smooth_diff[0] *= -Dtype(0.5);
	  }
	} else {
	  int count = top[0]->count();
	  caffe_gpu_hadamard_product<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, -Dtype(0.5), top_diff, w_norm_data, smooth_diff);
	}
      } else if (has_smooth_blobs_ && this->param_propagate_down_[0]) {
	// Gradient with respect to smooth_ param
	Dtype* smooth_diff = smooth_->mutable_gpu_diff();
	Dtype* smooth_cpu_diff = smooth_->mutable_cpu_diff();
	//caffe_gpu_set(smooth_->count(), Dtype(0), smooth_diff);
	if (unique_smooth_) {
	  int count = top[0]->count();
	  caffe_gpu_dot(count, w_norm_data, top_diff, smooth_cpu_diff);
	  smooth_cpu_diff[0] *= -Dtype(0.5);
	  LOG(INFO) << "gpu Smooth diff: " << smooth_cpu_diff[0];
	} else {
	  SmoothPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(channels_), CAFFE_CUDA_NUM_THREADS>>>(channels_, top_diff, num_, channels_,  w_norm_data, smooth_diff);
	  for (int i = 0; i < channels_; i++) {
	  LOG(INFO) << "gpu Smooth diff: " << smooth_cpu_diff[i];
         }
 }
      }
      CUDA_POST_KERNEL_CHECK;
    }

  //  if (!propagate_down[0]) {
  //    return;
  //  }
  //  const Dtype* top_diff = top[0]->gpu_diff();
  //  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  //  const int count = bottom[0]->count();
  //  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  //  // We'll output the mask to top[1] if it's of size >1.
  //  const bool use_top_mask = top.size() > 1;
  //  const int* mask = NULL;
  //  const Dtype* top_mask = NULL;
  //  switch (this->layer_param_.pooling_param().pool()) {
  //    case PoolingParameter_PoolMethod_MAX:
  //      if (use_top_mask) {
  //        top_mask = top[1]->gpu_data();
  //      } else {
  //        mask = max_idx_.gpu_data();
  //      }
  //      // NOLINT_NEXT_LINE(whitespace/operators)
  //      MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //          count, top_diff, mask, top_mask, top[0]->num(), channels_,
  //          height_, width_, pooled_height_, pooled_width_,
  //          kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
  //          bottom_diff);
  //      break;
  //    case PoolingParameter_PoolMethod_AVE:
  //      // NOLINT_NEXT_LINE(whitespace/operators)
  //      AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //          count, top_diff, top[0]->num(), channels_,
  //          height_, width_, pooled_height_, pooled_width_, kernel_h_,
  //          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
  //      break;
  //    case PoolingParameter_PoolMethod_STOCHASTIC:
  //      // NOLINT_NEXT_LINE(whitespace/operators)
  //      StoPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //          count, rand_idx_.gpu_data(), top_diff,
  //          top[0]->num(), channels_, height_, width_, pooled_height_,
  //          pooled_width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
  //          bottom_diff);
  //      break;
  //    default:
  //      LOG(FATAL) << "Unknown pooling method.";
  //  }
  //  CUDA_POST_KERNEL_CHECK;


  INSTANTIATE_LAYER_GPU_FUNCS(SmoothPoolingLayer);


}  // namespace caffe
