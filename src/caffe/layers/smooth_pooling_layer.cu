#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/smooth_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "iostream"
#include "stdio.h"

namespace caffe {


  template <typename Dtype>
    __global__ void SmoothPoolForward(const int nthreads, const Dtype* bottom_data, const int num, const int channels, const int dim, int* index_data, Dtype* value_data, const bool unique_smooth, const bool has_smooth_blobs, const Dtype z, const Dtype* smooth_data, Dtype* weight, Dtype* w_norm_data, Dtype* top_data) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const int n = index / channels;
	const int c = index % channels;
	int *U, *G, *L, *ind_tmp;
	U = index_data + 3 * dim * index;
	G = U + dim;
	L = G + dim;
	const Dtype* cur_bottom = bottom_data + dim * index;
	Dtype* v_tmp = value_data + dim * index;
	Dtype* w = weight + dim * index;
	Dtype* w_norm = w_norm_data + index;
	Dtype* o = top_data + index;
	double theta,  w_tmp;
	Dtype mu;
	if (has_smooth_blobs) {
	  if (unique_smooth) {
	    mu = smooth_data[0];
	  } else {
	    mu = smooth_data[c];
	  }
	} else {
	  if (unique_smooth) {
	    mu = smooth_data[n];
	  } else {
	    mu = smooth_data[index];
	  }
	}

	for (int i = 0; i < dim; i++) {
	  v_tmp[i] = Dtype(1) / (mu + Dtype(FLT_MIN)) * cur_bottom[i];
	  U[i] = i;
	}
	double s = 0, ds = 0, ro = 0, dro = 0;
	int n_U, n_G, n_L; 
	n_U = dim;
	while (n_U > Dtype(0)) {
	  int k = n_U-1;
	  n_G = 0; n_L =0;
	  ds = 0;
	  for(int i = 0; i < n_U; i++) {
	    if (v_tmp[U[i]] >= v_tmp[U[k]]) {
	      G[n_G++] = U[i];
	      ds += double(v_tmp[U[i]]);
	    } else {
	      L[n_L++] = U[i];
	    }
	  }
	  dro = double(n_G);

	  if ((s+ds) -(ro + dro) * double(v_tmp[U[k]]) < z) {
	    s += ds; ro += dro;
	    ind_tmp = U;
	    U = L;
	    n_U = n_L;
	    L = ind_tmp;
	  } else {
	    ind_tmp = U;
	    U = G;
	    n_U = n_G -1;
	    G = ind_tmp;
	  }
	}
	theta = (s-double(z)) / (ro + DBL_MIN);

	o[0] = Dtype(0);
	w_norm[0] = 0;
	for (int i = 0; i < dim; i++) {
	  w_tmp = double(v_tmp[i]) - theta;
	  w_tmp = w_tmp > 0 ? w_tmp : double(0);
	  w[i] = Dtype(w_tmp);
	  w_norm[0] += w[i] * w[i];
	  o[0] += cur_bottom[i] * Dtype(w[i]);
	}
	w_norm[0] *= -Dtype(0.5);
	o[0] += mu * w_norm[0];
      }
    }



  template <typename Dtype>
    void SmoothPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      int count = top[0]->count();
      const Dtype* smooth_data = smooth_->gpu_data();
      Dtype* weight_data =  weight_.mutable_gpu_data();
      Dtype* w_norm_data = w_norm_.mutable_gpu_data();
      Blob<int> index_set(3*num_, channels_, height_, width_);
      int* index_data = index_set.mutable_gpu_data();
      Blob<Dtype> value(num_, channels_, height_, width_);
      Dtype* value_data = value.mutable_gpu_data();
      SmoothPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, num_, channels_, dim_, 
	  index_data, value_data, unique_smooth_, has_smooth_blobs_, z_, smooth_data, weight_data, w_norm_data, top_data);
      CUDA_POST_KERNEL_CHECK;
    }
    
    template <typename Dtype> 
    __global__ void SmoothPoolBackwardBottom(const int nthreads, const Dtype* top_diff, const Dtype* weight, const int channels, const int dim,  Dtype* bottom_diff) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const int n = index / channels / dim;
	const int c = (index / dim) % channels;
	const int id = index % (dim);
	const Dtype* cur_top_diff = top_diff + n * channels + c;
	const Dtype* cur_weight = weight + (n * channels + c) * dim + id;
	Dtype* cur_bottom_diff = bottom_diff + (n * channels + c) * dim + id;
	cur_bottom_diff[0] = cur_top_diff[0] * cur_weight[0];

      }
    }


     template <typename Dtype> 
    __global__ void SmoothPoolBackwardUnique(const int nthreads, const Dtype* top_diff, const Dtype* w_norm, const int channels, Dtype* smooth_diff) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const Dtype* cur_top_diff = top_diff + index * channels;
	const Dtype* cur_w_norm = w_norm + index * channels;
	Dtype* cur_smooth_diff = smooth_diff + index;
	cur_smooth_diff[0] = 0;
	for (int i = 0; i < channels; i++) {
	  cur_smooth_diff[0] += cur_w_norm[i] * cur_top_diff[i];
	}
	//cur_smooth_diff[0] *= -Dtype(0.5);
      }
    }


  template <typename Dtype>
    __global__ void SmoothPoolBackward(const int nthreads, const Dtype* top_diff, const int num, const int channels, const Dtype* w_norm_data, Dtype* smooth_diff) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	const Dtype* cur_top_diff = top_diff + index;
	const Dtype* cur_w_norm = w_norm_data + index;

	for (int i = 0; i < num; i++) {
	  smooth_diff[index] += cur_top_diff[i*channels] * cur_w_norm[i*channels];
	}
	//smooth_diff[index] *= -Dtype(0.5); 
      }
    }

  template <typename Dtype>
    __global__ void caffe_gpu_hadamard_product(const int nthreads, const Dtype* a, const Dtype* b, Dtype* c) {
      CUDA_KERNEL_LOOP(index, nthreads) {
	c[index] = a[index] * b[index];
	}
    }



  template <typename Dtype>
    void SmoothPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* weight_data = weight_.gpu_data();
      const Dtype* w_norm_data = w_norm_.gpu_data();
      if (propagate_down[0]) {
	//Gradient with respect to bottom [0]
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	int count = bottom[0]->count();
	SmoothPoolBackwardBottom<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, weight_data, channels_, dim_,  bottom_diff);
      }
      if (!has_smooth_blobs_ && propagate_down[1]) {
	Dtype* smooth_diff = smooth_->mutable_gpu_diff();
	if (unique_smooth_) {
	  SmoothPoolBackwardUnique<Dtype><<<CAFFE_GET_BLOCKS(num_), CAFFE_CUDA_NUM_THREADS>>>(num_, top_diff, w_norm_data, channels_, smooth_diff); 
	} else {
	  int count = top[0]->count();
	  caffe_gpu_hadamard_product<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, w_norm_data, smooth_diff);
	}
      } else if (has_smooth_blobs_ && this->param_propagate_down_[0]) {
	// Gradient with respect to smooth_ param
	//caffe_gpu_set(smooth_->count(), Dtype(0), smooth_diff);
	if (unique_smooth_) {
	  int count = top[0]->count();
          Dtype* smooth_cpu_diff = smooth_->mutable_cpu_diff();
	  caffe_gpu_dot(count, w_norm_data, top_diff, smooth_cpu_diff);
	  //smooth_cpu_diff[0] *= -Dtype(0.5);
	} else {
          Dtype* smooth_diff = smooth_->mutable_gpu_diff();
	  SmoothPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(channels_), CAFFE_CUDA_NUM_THREADS>>>(channels_, top_diff, num_, channels_,  w_norm_data, smooth_diff);
	}
      }
      CUDA_POST_KERNEL_CHECK;
    }
  INSTANTIATE_LAYER_GPU_FUNCS(SmoothPoolingLayer);
}  // namespace caffe
