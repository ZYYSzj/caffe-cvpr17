#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/sigmoid_multi_label_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void SigmoidMultiLabelLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, const int label_size, const int bottom_dim, int* label_vector_data, Dtype* loss) {
  int num = nthreads / bottom_dim;
  CUDA_KERNEL_LOOP(index, nthreads) {
    // Set label_vector_data to zeros
    int n = index / bottom_dim;
    int c = index % bottom_dim;
    label_vector_data[n * bottom_dim + c] = 0;
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(index, nthreads) {
    if (index < num * label_size) {
      int sample_id = index / label_size;
      int label_id = index % label_size;
      int label_value = label[sample_id * label_size + label_id];
      if (label_value > 0) {
	label_vector_data[sample_id * bottom_dim + label_value - 1] = 1;
      }
    }
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / bottom_dim;
    int c = index % bottom_dim;
    int* cur_label_vector_data = label_vector_data + n * bottom_dim + c;
    const Dtype* cur_prob_data = prob_data + n * bottom_dim + c;
    loss[n * bottom_dim + c] = - cur_label_vector_data[0] * log(max(cur_prob_data[0], Dtype(kLOG_THRESHOLD))) - (1 - cur_label_vector_data[0]) * log(max(1 - cur_prob_data[0], Dtype(kLOG_THRESHOLD)));
  }
}

template <typename Dtype>
void SigmoidMultiLabelLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  int* label_vector_data = label_vector_.mutable_gpu_data();
  int n = prob_.num();
  int count = prob_.count();

  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  SigmoidMultiLabelLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, prob_data, label, label_size_, bottom_dim_, label_vector_data, loss_data);
  Dtype loss;
  caffe_gpu_asum(count, loss_data, &loss);
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  top[0]->mutable_cpu_data()[0] = loss / count;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SigmoidMultiLabelLossBackwardGPU(const int nthreads, const Dtype* prob,
          const int* label_vector_data, const int bottom_dim, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / bottom_dim;
    const int c = index % bottom_dim;
    const int label_value = static_cast<int>(label_vector_data[n * bottom_dim + c]);
    const Dtype prob_value = prob[n * bottom_dim + c];
    bottom_diff[n * bottom_dim + c] = -label_value * (1 - prob_value) + (1 - label_value) * prob_value;
  }
}

template <typename Dtype>
void SigmoidMultiLabelLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const int* label_vector_data = label_vector_.gpu_data();
    const int num = prob_.num(); 
    const int count = prob_.count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidMultiLabelLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, prob_data, label_vector_data, bottom_dim_,  bottom_diff);

    const Dtype loss_weight = top[0]->cpu_diff()[0] / num / bottom_dim_;
    caffe_gpu_scal(count, loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidMultiLabelLossLayer);

}  // namespace caffe
